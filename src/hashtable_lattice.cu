#include "hip/hip_runtime.h"
#include "cuda_utils.h"
#include "time_measure_util.h"
#include "hashtable_lattice.h"
#include <thrust/extrema.h>
#include <thrust/for_each.h>
#include <thrust/count.h>
#include <thrust/sort.h>
#include <map>
#include <cuco/static_map.cuh>
#include <unordered_set>

void update_permutation(const uint64_t* const keys, thrust::device_vector<uint32_t>& permutation)
{
    // temporary storage for keys
    thrust::device_vector<uint64_t> temp(permutation.size());

    // permute the keys with the current reordering
    thrust::device_ptr<const uint64_t> keys_ptr = thrust::device_pointer_cast(keys);
    thrust::gather(permutation.begin(), permutation.end(), keys_ptr, temp.begin());

    // stable_sort the permuted keys and update the permutation
    thrust::stable_sort_by_key(temp.begin(), temp.end(), permutation.begin());
}

template <typename T>
void apply_permutation(T* keys, thrust::device_vector<uint32_t>& permutation)
{
    // copy keys to temporary vector
    thrust::device_vector<T> temp(keys, keys + permutation.size());

    // permute the keys
    thrust::device_ptr<T> keys_ptr = thrust::device_pointer_cast(keys);
    thrust::gather(permutation.begin(), permutation.end(), temp.begin(), keys_ptr);
}

struct compute_max_min_coordinates {
    const int total_num_p; // includes batch size.
    const int d_pos;
    const int* const rem0;
    const int* const ranks;
    int* minimum_coordinates;
    int* maximum_coordinates;
    __device__ void operator()(const size_t i)
    {
        // TODOAA: Profile by transposing following two:
        // const int index_pt = i % total_num_p;
        // const int index_d = i / total_num_p;
        const int index_d = i % d_pos;
        const int index_pt = i / d_pos;
        assert (index_d < d_pos); // although lattice has dim = d_pos + 1, but only d_pos needs to be computed.
        const int current_rank = ranks[index_pt * (d_pos + 1) + index_d];
        const int current_rem0 = rem0[index_pt * (d_pos + 1) + index_d];
        int current_min, current_max;
        for (int r = 0; r != d_pos + 1; r++)
        {
            int pt_coordinate = current_rem0 + compute_canonical_simplex_point_coordinate(r, current_rank, d_pos);
            if (pt_coordinate != 0)
                pt_coordinate = floor_divisor(pt_coordinate, d_pos + 1);

            if (r == 0) {
                current_min = pt_coordinate;
                current_max = pt_coordinate;
            } else {
                current_min = min(current_min, pt_coordinate);
                current_max = max(current_max, pt_coordinate);
            }
        }
        atomicMin(&minimum_coordinates[index_d], current_min);
        atomicMax(&maximum_coordinates[index_d], current_max);
    }
};

struct count_bits {
    const int* minimum_coordinates;
    const int* maximum_coordinates;
    int* num_bits;
    __host__ __device__ void operator()(const size_t i)
    {
        num_bits[i] = (int) log2(maximum_coordinates[i] - minimum_coordinates[i] + 1) + 1;
    }
};

// rem0: [batch_size, num_positions, d_pos + 1]
// ranks: [batch_size, num_positions, d_pos + 1]
std::tuple<thrust::device_vector<int>, thrust::device_vector<int>> calculate_lattice_extents(
    const int batch_size, const int num_positions, const int d_pos, 
    const thrust::device_ptr<int> rem0, const thrust::device_ptr<int> ranks)
{
    thrust::device_vector<int> min_coordinate_per_pos(d_pos, std::numeric_limits<int>::max());
    thrust::device_vector<int> max_coordinate_per_pos(d_pos, std::numeric_limits<int>::min());

    const size_t num_workers = batch_size * num_positions * d_pos;
    compute_max_min_coordinates max_min_func({batch_size * num_positions, d_pos,
                        thrust::raw_pointer_cast(rem0), 
                        thrust::raw_pointer_cast(ranks), 
                        thrust::raw_pointer_cast(min_coordinate_per_pos.data()),
                        thrust::raw_pointer_cast(max_coordinate_per_pos.data())});

    thrust::for_each(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(0) + num_workers, max_min_func);

    const int d_encoded = d_pos + 2;
    thrust::device_vector<int> cumulative_num_bits_per_dim(d_encoded, 0);

    const int d_lattice = d_pos + 1;
    cumulative_num_bits_per_dim[d_encoded - 2] = (int) std::log2(d_lattice) + 1;
    cumulative_num_bits_per_dim[d_encoded - 1] = (int) std::log2(batch_size) + 1 + 1; // TODOAA: +1 for empty stencil

    // Compute number of bits required per each encoded dimension.
    count_bits count_bits_func({
                        thrust::raw_pointer_cast(min_coordinate_per_pos.data()),
                        thrust::raw_pointer_cast(max_coordinate_per_pos.data()),
                        thrust::raw_pointer_cast(cumulative_num_bits_per_dim.data())});

    thrust::for_each(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(0) + d_pos, count_bits_func);

    // Convert to cumulative:
    thrust::inclusive_scan(cumulative_num_bits_per_dim.begin(), cumulative_num_bits_per_dim.end(), cumulative_num_bits_per_dim.begin());
    std::vector<int> cumulative_num_bits_per_dim_h(cumulative_num_bits_per_dim.begin(), cumulative_num_bits_per_dim.end());
    int offset = 0;
    // print_vector(cumulative_num_bits_per_dim, "cumulative_num_bits_per_dim before");
    for (int i = 0; i < cumulative_num_bits_per_dim_h.size() - 1; i++)
    {
        const int pi = cumulative_num_bits_per_dim_h[i] / 64;
        const int pj = cumulative_num_bits_per_dim_h[i + 1] / 64;
        if (pi != pj)
            offset += pj * 64 - cumulative_num_bits_per_dim_h[i];
        cumulative_num_bits_per_dim_h[i] += offset;
    }
    cumulative_num_bits_per_dim_h.back() += offset;
    thrust::copy(cumulative_num_bits_per_dim_h.begin(), cumulative_num_bits_per_dim_h.end(), cumulative_num_bits_per_dim.begin());
    return {min_coordinate_per_pos, cumulative_num_bits_per_dim};
}

template<int NUM_VECS> 
hashtable_lattice<NUM_VECS>::hashtable_lattice(
    const int _batch_size, const int _num_positions, const int _d_pos, 
    const thrust::device_vector<int>& _min_coordinate_per_pos, const thrust::device_vector<int>& _cumulative_num_bits_per_dim) :
    batch_size(_batch_size), num_positions(_num_positions), d_pos(_d_pos), d_encoded(_d_pos + 2),
    min_coordinate_per_pos(_min_coordinate_per_pos), cumulative_num_bits_per_dim(_cumulative_num_bits_per_dim)
{
    number_of_required_bits = cumulative_num_bits_per_dim.back();
    // d_encoded = 1 + 1 + d_pos, where +1 for batch index, another +1 for reminder, 
    // rest d_pos for lattice point. Since lattice points sum to 0 so using d_pos instead of d_pos + 1.
}

template<int N> 
struct populate_hash_table {
    const int batch_size;
    const int num_positions;
    const int d_lattice;
    const int* const rem0;
    const int* const ranks;
    const int* const cumulative_num_bits;
    const int* const min_coordinate;
    typename cuco::static_map<encoded_lattice_pt<N>, VALUE_TYPE>::device_mutable_view hasht_view;
    __device__ bool operator()(const size_t i)
    {
        const int reminder = i % d_lattice;
        const int index_pt = (i / d_lattice) % num_positions;
        const int batch_index = i / (d_lattice * num_positions);
        const int start_index = (batch_index * num_positions + index_pt) * d_lattice;
        const encoded_lattice_pt<N> encoded_pt = encode_point<encoded_lattice_pt<N>>(
            cumulative_num_bits, min_coordinate, rem0, ranks, batch_index, start_index, reminder, d_lattice);
        const auto inserted = hasht_view.insert(cuco::make_pair(encoded_pt, (VALUE_TYPE) i), hash_encoded_lattice_pt<N>{});
        // auto [iter, inserted] = hasht_view.insert_and_find<HASH_FUNCTION>(cuco::pair(encoded_pt, (VALUE_TYPE) i));
        // if (!inserted)
        //     iter->second = min(iter->second, (VALUE_TYPE) i);
        return inserted;
    }
};

template<int NUM_VECS> 
void hashtable_lattice<NUM_VECS>::add_points_to_lattice(const thrust::device_ptr<int> rem0, const thrust::device_ptr<int> ranks)
{
    // Create hashtable with maximum capacity as each point can splat to d_pos + 1 many points.
    cuco::empty_key<encoded_lattice_pt<NUM_VECS>> empty_key_sentinel(encoded_lattice_pt<NUM_VECS>::create_empty_stencil());
    hash_table = std::make_unique<cuco::static_map<encoded_lattice_pt<NUM_VECS>, VALUE_TYPE>>(2 * batch_size * num_positions * (d_pos + 1), empty_key_sentinel, empty_value_sentinel);

    populate_hash_table<NUM_VECS> populate_hashtable_func({
        batch_size, num_positions, d_pos + 1, 
        thrust::raw_pointer_cast(rem0),
        thrust::raw_pointer_cast(ranks),
        thrust::raw_pointer_cast(cumulative_num_bits_per_dim.data()),
        thrust::raw_pointer_cast(min_coordinate_per_pos.data()),
        hash_table->get_device_mutable_view()});
    
    const size_t num_workers = batch_size * num_positions * (d_pos + 1);
    num_lattice_points = thrust::count_if(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(0) + num_workers, populate_hashtable_func);

    std::cout<<"Populated hash table. Size: "<<num_lattice_points<<", capacity: "<<hash_table->get_capacity()<<"\n";
}

template<int N> 
struct make_values_consecutive {
    const encoded_lattice_pt<N>* const keys;
    typename cuco::static_map<encoded_lattice_pt<N>, VALUE_TYPE>::device_view hasht_view;
    __device__ void operator()(const size_t i)
    {
        auto iter = hasht_view.find(keys[i], hash_encoded_lattice_pt<N>{});
        iter->second = i;
    }
};

template<int NUM_VECS> 
void hashtable_lattice<NUM_VECS>::make_values_contiguous()
{
    // Now make values of hashtable lie in {0, 1, ..., num_lattice_points - 1} thus allowing all subsequent operations on the lattice use less memory.
    // For that sort hash table values (with keys as keys) and then use index of this sorted array to compute new index. Afterwards call thrust struct to
    // update hash table values by doing find() on this sorted keys and replace by their corresponding values. 
    thrust::device_vector<encoded_lattice_pt<NUM_VECS>> keys;
    thrust::device_vector<VALUE_TYPE> values;
    std::tie(keys, values) = this->get_hashtable_entries();

    thrust::sort_by_key(values.begin(), values.end(), keys.begin());
    make_values_consecutive<NUM_VECS> make_values_consecutive_func({
        thrust::raw_pointer_cast(keys.data()),
        hash_table->get_device_view()});

    thrust::for_each(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(0) + keys.size(), make_values_consecutive_func);
}

template<int N> 
struct decode_points_from_keys {
    const int d_lattice;
    const int* const cumulative_num_bits;
    const int* const min_coordinate;
    const encoded_lattice_pt<N>* const encoded_keys;
    int* decoded_lattice_points;
    __device__ void operator()(const size_t index_pt)
    {
        decode_point<encoded_lattice_pt<N>>(cumulative_num_bits, min_coordinate, index_pt, d_lattice, encoded_keys[index_pt], decoded_lattice_points);
    }
};

template<int NUM_VECS> 
std::tuple<thrust::device_vector<encoded_lattice_pt<NUM_VECS>>, thrust::device_vector<VALUE_TYPE>> hashtable_lattice<NUM_VECS>::get_hashtable_entries() const
{
    thrust::device_vector<encoded_lattice_pt<NUM_VECS>> keys(num_lattice_points);
    thrust::device_vector<VALUE_TYPE> values(num_lattice_points);

    auto [key_end, value_end] = hash_table->retrieve_all(keys.begin(), values.begin());
    assert(std::distance(keys.begin(), key_end) == keys.size());
    assert(std::distance(values.begin(), value_end) == keys.size());
    return {keys, values};
}

template<int NUM_VECS> 
std::tuple<thrust::device_vector<int>, thrust::device_vector<encoded_lattice_pt<NUM_VECS>>, thrust::device_vector<VALUE_TYPE>> 
    hashtable_lattice<NUM_VECS>::get_valid_lattice_points_and_indices() const
{
    thrust::device_vector<encoded_lattice_pt<NUM_VECS>> keys;
    thrust::device_vector<VALUE_TYPE> values;
    std::tie(keys, values) = this->get_hashtable_entries();

    thrust::sort_by_key(values.begin(), values.end(), keys.begin());
    thrust::device_vector<int> lattice_points(num_lattice_points * (d_pos + 1));
    decode_points_from_keys<NUM_VECS> decode_points_from_keys_func({
        d_pos + 1, 
        thrust::raw_pointer_cast(cumulative_num_bits_per_dim.data()),
        thrust::raw_pointer_cast(min_coordinate_per_pos.data()),
        thrust::raw_pointer_cast(keys.data()),
        thrust::raw_pointer_cast(lattice_points.data())});
    
    const size_t num_workers = num_lattice_points;
    thrust::for_each(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(0) + num_workers, decode_points_from_keys_func);
    return {lattice_points, keys, values};
}

template<int N> 
struct get_splatting_indices_kernel {
    const int batch_size;
    const int num_positions;
    const int d_lattice;
    const int* const rem0;
    const int* const ranks;
    const int* const cumulative_num_bits;
    const int* const min_coordinate;
    typename cuco::static_map<encoded_lattice_pt<N>, VALUE_TYPE>::device_view hasht_view;
    VALUE_TYPE* splatting_indices;
    __device__ bool operator()(const size_t i)
    {
        const int reminder = i % d_lattice;
        const int index_pt = (i / d_lattice) % num_positions;
        const int batch_index = i / (d_lattice * num_positions);
        const int start_index = (batch_index * num_positions + index_pt) * d_lattice;
        const encoded_lattice_pt<N> encoded_pt = encode_point<encoded_lattice_pt<N>>(
            cumulative_num_bits, min_coordinate, rem0, ranks, batch_index, start_index, reminder, d_lattice);
        const auto iter = hasht_view.find(encoded_pt, hash_encoded_lattice_pt<N>{});
        assert(iter != hasht_view.end()); // key should exist.
        splatting_indices[i] = iter->second;
    }
};

// output should be preallocated to size batch_size * num_positions * (d_pos + 1).
template<int NUM_VECS> 
void hashtable_lattice<NUM_VECS>::get_splatting_indices(
    const thrust::device_ptr<int> rem0, const thrust::device_ptr<int> ranks, thrust::device_ptr<VALUE_TYPE> output) const
{
    // Each input point splats to exactly d_pos + 1 many lattice points.
    const size_t num_workers = batch_size * num_positions * (d_pos + 1);
    get_splatting_indices_kernel<NUM_VECS> get_splatting_indices_func({batch_size, num_positions, d_pos + 1, 
        thrust::raw_pointer_cast(rem0),
        thrust::raw_pointer_cast(ranks),        
        thrust::raw_pointer_cast(cumulative_num_bits_per_dim.data()),
        thrust::raw_pointer_cast(min_coordinate_per_pos.data()),
        hash_table->get_device_view(),
        thrust::raw_pointer_cast(output)});
    
    thrust::for_each(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(0) + num_workers, get_splatting_indices_func);
}

template <int N>
struct struct_of_arrays
{
    std::array<uint64_t*, N> data;
};

template <int N>
struct get_encoded_points_kernel {
    const int batch_size;
    const int num_positions;
    const int d_lattice;
    const int* const rem0;
    const int* const ranks;
    const int* const cumulative_num_bits;
    const int* const min_coordinate;
    // KEY_TYPE* encoded_points;
    struct_of_arrays<N> encoded_points;
    uint32_t* sorting_order;
    __device__ bool operator()(const size_t i)
    {
        const int reminder = i % d_lattice;
        const int index_pt = (i / d_lattice) % num_positions;
        const int batch_index = i / (d_lattice * num_positions);
        const int start_index = (batch_index * num_positions + index_pt) * d_lattice;
        const encoded_lattice_pt<N> encoded_pt = encode_point<encoded_lattice_pt<N>>(
            cumulative_num_bits, min_coordinate, rem0, ranks, batch_index, start_index, reminder, d_lattice);
        // writing encoded_points in the following order to ease sorting:
        // batch_index, reminder, point_index
        const size_t out_index = (batch_index * num_positions * d_lattice) + (reminder * num_positions) + index_pt;
        for (int i = 0; i != N; ++i)
            encoded_points.data[i][out_index] = encoded_pt.data[i];
        sorting_order[out_index] = i;
    }
};

template <int N>
struct mark_first_unique_lattice_pt {
    const struct_of_arrays<N> encoded_points;
    uint32_t* is_unique;
    __device__ bool operator()(const size_t i)
    {
        assert(i > 0);
        bool cur_is_unique = false;
        for (int s = 0; s != N; ++s)
        {
            if(encoded_points.data[s][i] != encoded_points.data[s][i - 1])
            {
                cur_is_unique = true;
                break;
            }
        }
        if (cur_is_unique)
            is_unique[i] = 1;
    }
};

// output should be preallocated to size batch_size * num_positions * (d_pos + 1).
template<int NUM_VECS>
int hashtable_lattice<NUM_VECS>::get_splatting_indices_direct(
    const thrust::device_ptr<int> rem0, const thrust::device_ptr<int> ranks, thrust::device_ptr<VALUE_TYPE> output)
{
    // Each input point splats to exactly d_pos + 1 many lattice points.
    const size_t num_workers = batch_size * num_positions * (d_pos + 1);

    struct_of_arrays<NUM_VECS> encoded_pts;

    for (int i = 0; i != NUM_VECS;  i++)
        hipMalloc(&encoded_pts.data[i], num_workers * sizeof(uint64_t));

    thrust::device_vector<uint32_t> sorting_order(num_workers);
    {
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME2("[get_splatting_indices_direct] get_encoded_points_kernel");
        get_encoded_points_kernel<NUM_VECS> get_encoded_points_func({batch_size, num_positions, d_pos + 1, 
            thrust::raw_pointer_cast(rem0),
            thrust::raw_pointer_cast(ranks),        
            thrust::raw_pointer_cast(cumulative_num_bits_per_dim.data()),
            thrust::raw_pointer_cast(min_coordinate_per_pos.data()),
            encoded_pts,
            thrust::raw_pointer_cast(sorting_order.data())});

        thrust::for_each(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(0) + num_workers, get_encoded_points_func);
    }

    // // debugging:
    // std::vector<std::vector<uint64_t>> host_points(NUM_VECS);
    // for (int i = 0; i != NUM_VECS;  i++)
    // {
    //     host_points[i] = std::vector<uint64_t>(num_workers);
    //     hipMemcpy(host_points[i].data(), encoded_pts.data[i], num_workers * sizeof(uint64_t), hipMemcpyDeviceToHost);
    // }

    // std::cout<<"copied\n";
    // std::unordered_set<encoded_lattice_pt<NUM_VECS>, hash_encoded_lattice_pt<NUM_VECS>> host_set;
    // for (int i = 0; i != num_workers; i++)
    // {
    //     encoded_lattice_pt<NUM_VECS> current_pt;
    //     for (int j = 0; j != NUM_VECS;  j++)
    //         current_pt.data[j] = host_points[j][i];
    //     host_set.emplace(current_pt);
    // }
    // std::cout<<"inserted\n";
    // std::cout<<"Debugging. Number of unique lattice points: "<<host_set.size()<<"\n";
    // // debugging end.

    {
        std::string message = "[get_splatting_indices_direct] sort encoded points, encoding size: ";
        message += std::to_string(NUM_VECS) + " x 8bytes, ";
        MEASURE_CUMULATIVE_FUNCTION_EXECUTION_TIME2(message);

        // sort from least significant key to most significant keys
        thrust::device_vector<uint32_t> permutation(num_workers);
        thrust::sequence(permutation.begin(), permutation.end());
        for (int i = 0; i != NUM_VECS;  i++)
            update_permutation(encoded_pts.data[i], permutation);

        for (int i = 0; i != NUM_VECS;  i++)
            apply_permutation(encoded_pts.data[i], permutation);
        apply_permutation(thrust::raw_pointer_cast(sorting_order.data()), permutation);
    }

    thrust::device_vector<uint32_t> is_unique_pt(num_workers, 0);
    mark_first_unique_lattice_pt<NUM_VECS> mark_first_unique_lattice_pt_func({
        encoded_pts,
        thrust::raw_pointer_cast(is_unique_pt.data())
    });

    thrust::for_each(thrust::make_counting_iterator<size_t>(0) + 1, thrust::make_counting_iterator<size_t>(0) + num_workers,
                    mark_first_unique_lattice_pt_func);

    // convert to index:
    thrust::inclusive_scan(is_unique_pt.begin(), is_unique_pt.end(), is_unique_pt.begin());

    num_lattice_points = is_unique_pt.back() + 1;
    for (int i = 0; i != NUM_VECS;  i++)
        hipFree(encoded_pts.data[i]);
    
    thrust::scatter(is_unique_pt.begin(), is_unique_pt.end(), sorting_order.begin(), output);
    return num_lattice_points;
}

template<int N>
struct compute_blur_neighbours_kernel {
    const int num_pts;
    const int d_lattice;
    const encoded_lattice_pt<N>* const encoded_keys;
    const VALUE_TYPE* const values;
    const int* const cumulative_num_bits;
    typename cuco::static_map<encoded_lattice_pt<N>, VALUE_TYPE>::device_view hasht_view;
    VALUE_TYPE* const n1;
    VALUE_TYPE* const n2;
    __device__ void operator()(const size_t i)
    {
        // consecutive threads will operate on consecutive points and same direction. // TODOAA: benchmark.
        // const int index_pt = i % num_pts;
        // const int direction = i / num_pts;
        // consecutive threads will operate on same point with different direction.
        const int direction = i % d_lattice;
        const int index_pt = i / d_lattice;
        const encoded_lattice_pt<N> self_point = encoded_keys[index_pt];
        encoded_lattice_pt<N> neighbour_point_plus(self_point);
        encoded_lattice_pt<N> neighbour_point_minus(self_point);
        assert(neighbour_point_plus == self_point);
        assert(neighbour_point_minus == self_point);
        bool plus_overflow, minus_overflow;
        compute_neighbour_encoding<encoded_lattice_pt<N>>(cumulative_num_bits, d_lattice, self_point, direction, 
            neighbour_point_plus, neighbour_point_minus, plus_overflow, minus_overflow);

        const VALUE_TYPE output_index = direction * num_pts + values[index_pt];

        if (!plus_overflow)
        {
            const auto iter = hasht_view.find(neighbour_point_plus, hash_encoded_lattice_pt<N>{});
            if(iter != hasht_view.end())
                n1[output_index] = iter->second;
        }
        if (!minus_overflow)
        {
            const auto iter = hasht_view.find(neighbour_point_minus, hash_encoded_lattice_pt<N>{});
            if(iter != hasht_view.end())
                n2[output_index] = iter->second;
        }
    }
};

//     blur_n1: [d_lattice, num_splatted_points] should be preallocated.
//     blur_n2: [d_lattice, num_splatted_points] should be preallocated.
template<int NUM_VECS>
void hashtable_lattice<NUM_VECS>::compute_blur_neighbours(thrust::device_ptr<VALUE_TYPE> blur_n1, thrust::device_ptr<VALUE_TYPE> blur_n2) const
{
    size_t num_workers = (d_pos + 1) * num_lattice_points;
    thrust::fill(blur_n1, blur_n1 + ((d_pos + 1) * num_lattice_points), num_lattice_points); // values = num_lattice_points indicate no neighbour found.
    thrust::fill(blur_n2, blur_n2 + ((d_pos + 1) * num_lattice_points), num_lattice_points);
    thrust::device_vector<encoded_lattice_pt<NUM_VECS>> keys;
    thrust::device_vector<VALUE_TYPE> values;
    std::tie(keys, values) = this->get_hashtable_entries();

    compute_blur_neighbours_kernel<NUM_VECS> compute_blur_neighbours_func({
        num_lattice_points, d_pos + 1, 
        thrust::raw_pointer_cast(keys.data()),
        thrust::raw_pointer_cast(values.data()),
        thrust::raw_pointer_cast(cumulative_num_bits_per_dim.data()),
        hash_table->get_device_view(),
        thrust::raw_pointer_cast(blur_n1), 
        thrust::raw_pointer_cast(blur_n2)});

    thrust::for_each(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(0) + num_workers, compute_blur_neighbours_func);
}

template<int N>
struct compute_blur_neighbours_direct_kernel {
    const int num_lattice_points;
    const int d_lattice;
    const int* const rem0;
    const int* const ranks;
    const int* const splatting_table;
    VALUE_TYPE* const n1;
    VALUE_TYPE* const n2;
    __device__ void operator()(const size_t i)
    {
        const int self_reminder = i % d_lattice;
        const int start_index = i - self_reminder;
        const int self_splat_index = splatting_table[i];

        for(int other_reminder = self_reminder - 1; other_reminder <= self_reminder + 1; ++other_reminder)
        {
            if (other_reminder == self_reminder)
                continue;

            const int other_reminder_pos = positive_modulo(other_reminder, d_lattice);
            int blur_direction = 0;
            for (int index_d = 0; index_d != d_lattice; index_d++)
            {
                const int cur_rank = ranks[start_index + index_d];
                const int self_pt_coordinate = compute_canonical_simplex_point_coordinate(self_reminder, cur_rank, d_lattice - 1);
                const int other_pt_coordinate = compute_canonical_simplex_point_coordinate(other_reminder_pos, cur_rank, d_lattice - 1);
                const int diff = other_pt_coordinate - self_pt_coordinate;
                if (abs(diff) == 1)
                    continue;
                blur_direction = index_d;
                break;
            }
            const int other_splat_index = splatting_table[start_index + other_reminder_pos];
            const VALUE_TYPE output_index = blur_direction * num_lattice_points + self_splat_index;
            // if (i < 1000)
                // printf("self_pt_coordinate: %d, other_pt_coordinate: %d, diff: %d, self_splat_index: %d, other_splat_index: %d, self_reminder: %d, other_reminder: %d, other_reminder_pos: %d\n", 
                // self_pt_coordinate, other_pt_coordinate, diff, self_splat_index, other_splat_index, self_reminder, other_reminder, other_reminder_pos);
                // printf("self_splat_index: %d, other_splat_index: %d, self_reminder: %d, other_reminder_pos: %d, output_index, : %d\n", 
                //         self_splat_index, other_splat_index, self_reminder, other_reminder_pos, output_index);
            if (other_reminder < self_reminder)
                n1[output_index] = other_splat_index;
            else
                n2[output_index] = other_splat_index;
        }
    }
};

// rem0: [batch_size, num_positions, d_pos + 1]
// ranks: [batch_size, num_positions, d_pos + 1]
// splatting_table: [batch_size, num_positions, (d_pos + 1)]
// blur_n1: [d_lattice, num_splatted_points] should be preallocated.
// blur_n2: [d_lattice, num_splatted_points] should be preallocated.
template<int NUM_VECS>
void hashtable_lattice<NUM_VECS>::compute_blur_neighbours_direct(
    const thrust::device_ptr<int> rem0, const thrust::device_ptr<int> ranks, thrust::device_ptr<VALUE_TYPE> splatting_table, 
    thrust::device_ptr<VALUE_TYPE> blur_n1, thrust::device_ptr<VALUE_TYPE> blur_n2) const
{
    const size_t num_workers = batch_size * num_positions * (d_pos + 1);
    thrust::fill(blur_n1, blur_n1 + (num_lattice_points * (d_pos + 1)), num_lattice_points); // values = num_lattice_points indicate no neighbour found.
    thrust::fill(blur_n2, blur_n2 + (num_lattice_points * (d_pos + 1)), num_lattice_points);

    compute_blur_neighbours_direct_kernel<NUM_VECS> compute_blur_neighbours_func({
        num_lattice_points, d_pos + 1, 
        thrust::raw_pointer_cast(rem0),
        thrust::raw_pointer_cast(ranks),
        thrust::raw_pointer_cast(splatting_table),
        thrust::raw_pointer_cast(blur_n1), 
        thrust::raw_pointer_cast(blur_n2)});

    thrust::for_each(thrust::make_counting_iterator<size_t>(0), thrust::make_counting_iterator<size_t>(0) + num_workers, compute_blur_neighbours_func);
}


// std::vector<int> hashtable_lattice<NUM_VECS>::compute_all_lattice_points_slow(const thrust::device_ptr<int> rem0_d, const thrust::device_ptr<int> ranks_d) const
// {
//     std::vector<int> rem0(rem0_d, rem0_d + batch_size * num_positions * (d_pos + 1));
//     std::vector<int> ranks(ranks_d, ranks_d + batch_size * num_positions * (d_pos + 1));
//     std::vector<int> min_c(min_coordinate_per_pos.begin(), min_coordinate_per_pos.end());
//     std::vector<int> cumulative_num_bits(cumulative_num_bits_per_dim.begin(), cumulative_num_bits_per_dim.end());
//     std::vector<int> output;
//     for(int index_pt = 0; index_pt != batch_size * num_positions; ++index_pt)
//     {
//         const int batch_index = index_pt / num_positions;
//         for (int r = 0; r != d_pos + 1; r++)
//         {
//             std::cout<<"batch_index: "<<batch_index<<", index_pt: "<<index_pt<<", r: "<<r<<"\n\t";
//             output.push_back(batch_index);
//             uint32_t packedNumber = 0;
//             int shift = 0;
//             for(int index_d = 0; index_d != d_pos; ++index_d)
//             {
//                 const int current_rank = ranks[index_pt * (d_pos + 1) + index_d];
//                 const int current_rem0 = rem0[index_pt * (d_pos + 1) + index_d];
//                 int coord = current_rem0 + compute_canonical_simplex_point_coordinate(r, current_rank, d_pos);
//                 std::cout<<" "<<coord;
//                 output.push_back(coord);

//                 if (coord != 0)
//                     coord = floor_divisor(coord, d_pos + 1);
//                 coord -= min_c[index_d];
//                 std::cout<<"("<<coord<<")";
//                 assert(coord >= 0);
//                 // Pack the number by shifting it and combining with the packedNumber
//                 packedNumber |= coord << shift;
//                 shift = cumulative_num_bits[index_d];
//             }
//             packedNumber |= r << shift;
//             packedNumber |= batch_index << cumulative_num_bits[d_pos];
//             std::cout<<"\n\tkey: "<<packedNumber<<"\n";
//         }
//     }
//     return output;
// }

// size_t hashtable_lattice<NUM_VECS>::test_lattice_point_encoder(const thrust::device_ptr<int> rem0_d, const thrust::device_ptr<int> ranks_d) const
// {
//     print_vector(cumulative_num_bits_per_dim, "cumulative_num_bits_per_dim");
//     print_vector(min_coordinate_per_pos, "min_coordinate_per_pos");
//     auto print_std_vector = [](const std::vector<int>& vec, const char* name) {
//         std::cout<<name<<": ";
//         for (const auto& element : vec) {
//             std::cout << element << " ";
//         }
//         std::cout << std::endl;
//     };
//     std::map<KEY_TYPE, std::pair<VALUE_TYPE, std::vector<int>>> map;
//     std::vector<int> hash_collisions(2 * batch_size * num_positions * (d_pos + 1), 0);
//     std::vector<int> rem0(rem0_d, rem0_d + batch_size * num_positions * (d_pos + 1));
//     std::vector<int> ranks(ranks_d, ranks_d + batch_size * num_positions * (d_pos + 1));
//     std::vector<int> min_c(min_coordinate_per_pos.begin(), min_coordinate_per_pos.end());
//     std::vector<int> cumulative_num_bits(cumulative_num_bits_per_dim.begin(), cumulative_num_bits_per_dim.end());
//     size_t num_unique_points = 0;
//     for(int index_pt = 0; index_pt != batch_size * num_positions; ++index_pt)
//     {
//         const int batch_index = index_pt / num_positions;
//         for (int r = 0; r != d_pos + 1; r++)
//         {
//             KEY_TYPE packedNumber;
//             int start_bit = 0;
//             int end_bit = cumulative_num_bits[0];
//             std::vector<int> current_point;
//             for(int index_d = 0; index_d != d_pos + 2; ++index_d)
//             {
//                 if (index_d < d_pos)
//                 {
//                     const int current_rank = ranks[index_pt * (d_pos + 1) + index_d];
//                     const int current_rem0 = rem0[index_pt * (d_pos + 1) + index_d];
//                     int coord = current_rem0 + compute_canonical_simplex_point_coordinate(r, current_rank, d_pos);
//                     current_point.push_back(coord);
//                     if (coord != 0)
//                         coord = floor_divisor(coord, d_pos + 1);
//                     coord -= min_c[index_d];
//                     assert(coord >= 0);
//                     packedNumber.put_number(start_bit, end_bit, coord);
//                 }
//                 else if (index_d == d_pos)
//                     packedNumber.put_number(start_bit, end_bit, r);
//                 else
//                     packedNumber.put_number(start_bit, end_bit, batch_index);
//                 start_bit = end_bit;
//                 end_bit = cumulative_num_bits[index_d + 1];
//             }
//             HASH_FUNCTION hasher;
//             const auto hash = hasher(packedNumber) % hash_collisions.size();
//             hash_collisions[hash]++;
//             const auto [it, success] = map.emplace(packedNumber, std::make_pair(num_unique_points, current_point));
//             if(success)
//                 num_unique_points++;
//             else
//             {
//                 // check if pre-existing point has same point coordinates.
//                 std::vector<int> existing_point = std::get<1>(it->second);
//                 if (existing_point != current_point)
//                 {
//                     // std::cout<<"\nCoordinates mis-match for encoded key: "<<packedNumber<<"\n";
//                     print_std_vector(current_point, "current_point");
//                     print_std_vector(existing_point, "existing_point");
//                     throw std::runtime_error("exiting.");
//                 }
//             }
//         }
//     }
//     std::sort(hash_collisions.begin(), hash_collisions.end(), std::greater<int>());
//     hash_collisions.resize(100);
//     print_std_vector(hash_collisions, "hash_collisions");
//     return num_unique_points;
// }

template class hashtable_lattice<1>;
template class hashtable_lattice<2>;
// template class hashtable_lattice<3>;
template class hashtable_lattice<4>;
// template class hashtable_lattice<5>;
template class hashtable_lattice<6>;
// template class hashtable_lattice<7>;
template class hashtable_lattice<8>;
// template class hashtable_lattice<9>;